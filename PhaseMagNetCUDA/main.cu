
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include "LinkedList.cuh"
#include "pmncudautils.cuh"
#include "cudafuncs.cuh"
#include "PhaseMagNetCUDA.cuh"
#include "readdataset.cuh"

void buildNetwork(PhaseMagNetCUDA& net) {
    /* Input Layer */
    MatrixDim in_mdim(28, 28, sizeof(DTYPE), 1);
    LayerParams input(LayerType::input, ActivationType::relu, in_mdim);
    net.addLayer(input);
    /* Conv1 */
    ConvParams conv1;
    MatrixDim fDim(5, 5, sizeof(DTYPE), 1);
    conv1.filterDim = fDim;
    conv1.pad = 2;
    conv1.stride = 1;
    conv1.numFilters = 6;
    MatrixDim convMdim(conv1.getNextActDim(in_mdim, sizeof(DTYPE)));
    LayerParams convLayer(LayerType::conv, ActivationType::relu, convMdim, conv1);
    net.addLayer(convLayer);
    /* Average Pooling */
    ConvParams avgPoolParams;
    MatrixDim tmp(2, 2, sizeof(DTYPE), 6);
    avgPoolParams.filterDim = tmp;
    avgPoolParams.pad = 0;
    avgPoolParams.stride = 2;
    avgPoolParams.numFilters = 6;
    MatrixDim avgPoolMdim(avgPoolParams.getNextActDim(convMdim, sizeof(DTYPE)));
    LayerParams avgPoolLayer(LayerType::avgpool, ActivationType::relu, avgPoolMdim, avgPoolParams);
    net.addLayer(avgPoolLayer);
    /* FC 1 */
    MatrixDim mid1_mdim(1, 120, sizeof(DTYPE));
    LayerParams mid1(LayerType::fc, ActivationType::relu, mid1_mdim);
    net.addLayer(mid1);
    /* FC 2 */
    MatrixDim mid2_mdim(1, 84, sizeof(DTYPE));
    LayerParams mid2(LayerType::fc, ActivationType::relu, mid2_mdim);
    net.addLayer(mid2);
    /* FC 2 >>> OUTPUT <<< */
    MatrixDim out_mdim(1, 10, sizeof(DTYPE));
    LayerParams output(LayerType::fc, ActivationType::softmax, out_mdim);
     net.addLayer(output);
     net.initialize();
}

int main()
{
    int n_ims_train = 50000;
    int n_ims_test = 10000;
    int image_size = 784;
    PhaseMagNetCUDA net;
    //buildNetwork(net);
    net.load("convpmnn7.txt");

    printf("Loading Data...\n");
    uchar** imdata = read_mnist_images("..\\..\\..\\..\\mnist\\train-images-idx3-ubyte", n_ims_train, image_size);
    uchar* ladata = read_mnist_labels("..\\..\\..\\..\\mnist\\train-labels-idx1-ubyte", n_ims_train);
    uchar** imdata_test = read_mnist_images("..\\..\\..\\..\\mnist\\t10k-images-idx3-ubyte", n_ims_test, image_size); //t10k-images-idx3-ubyte // ann_a_advclp_0.2eps-ubyte
    uchar* ladata_test = read_mnist_labels("..\\..\\..\\..\\mnist\\t10k-labels-idx1-ubyte", n_ims_test);
    printf("Finished Loading Data.\n");

    float lrnRate = 0.0005f;
    for (int i = 1; i <= 1; ++i) {
        printf("Epoch: %d\n", i);
        float acc = net.evaluate(/*n_ims_test*/ 10000, imdata_test, ladata_test, /* verbose */ true);
        printf("Acc: %4.2f \n", acc * 100.0);
        net.train(/* n_ims_train */ 50000, imdata, ladata, /* */ lrnRate, /* verbose */ true);
        printf("\n");
    }
    net.save(".\\convpmnn7.txt");
    net.free();
    // printf("index: %d %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f true: %d\n", i,  o[0], o[1], o[2], o[3], o[4], o[5], o[6], o[7], o[8], o[9], ladata[i]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }

    return 0;
}